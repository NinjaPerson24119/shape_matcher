#include "hip/hip_runtime.h"
/**
 * @author Nicholas Wengel
 */ 

#include <au_vision/shape_analysis/gpu_util_kernels.h>

namespace au_vision
{

//Forward declarations
__global__ void inRange_device(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSzb dst, int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2);
__global__ void simpleEdgeDetect_device(unsigned short* grayMask, unsigned char* binaryMask, int rows, int cols);
__device__ bool sameColor_device(unsigned short* grayMask, unsigned char* binaryMask, int rows, int cols, int idx, int movX, int movY);

//Kernel credits for inRange: https://github.com/opencv/opencv/issues/6295
void callInRange_device(const cv::cuda::GpuMat &src, const cv::Scalar &lowerb, const cv::Scalar &upperb,
                 cv::cuda::GpuMat &dst) {
   //Max block size of 1024 (as per spec)
   int m = global_threadsPerBlock;
   if(m > 32)
   {
     m = 32;
   }

  //Initialize the output (this was fixed from the sourced page)
  dst.create(src.rows, src.cols, CV_8UC1);

  int numRows = src.rows, numCols = src.cols;
  if (numRows == 0 || numCols == 0) return;
  // Attention! Cols Vs. Rows are reversed
  const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
  const dim3 blockSize(m, m, 1);

  inRange_device<<<gridSize, blockSize>>>(src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1],
                                          lowerb[2], upperb[2]);
  hipDeviceSynchronize();

  gpuErrorCheck( hipGetLastError() ); //Verify that all went OK
}

void callSimpleEdgeDetect_device(unsigned short* grayMask, unsigned char* binaryMask, int rows, int cols)
{
    //Each thread will sum a grid square
    int blocks = std::ceil((double)(rows * cols) / 32);
    int threadsPerBlock = 32;
  
    simpleEdgeDetect_device<<<blocks, threadsPerBlock>>>(grayMask, binaryMask, rows, cols);
    hipDeviceSynchronize();  // Block until kernel queue finishes
    gpuErrorCheck( hipGetLastError() ); //Verify that all went OK
}

__global__ void inRange_device(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSzb dst, int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= src.cols || y >= src.rows) return;

  uchar3 v = src(y, x);
  if (v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
    dst(y, x) = 255;
  else
    dst(y, x) = 0;
}

__global__ void simpleEdgeDetect_device(unsigned short* grayMask, unsigned char* binaryMask, int rows, int cols)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if(idx < rows * cols)
  {
    bool isWhite = false;
    
    //If the color is not surrounded by the same color , set it to white 
    //(left, right, up, down)
    
    if(!sameColor_device(grayMask, binaryMask, rows, cols, idx, -1, 0))
    {
      isWhite = true;
    }
    if(!sameColor_device(grayMask, binaryMask, rows, cols, idx, 1, 0))
    {
      isWhite = true;
    }
    if(!sameColor_device(grayMask, binaryMask, rows, cols, idx, 0, -1))
    {
      isWhite = true;
    }
    if(!sameColor_device(grayMask, binaryMask, rows, cols, idx, 0, 1))
    {
      isWhite = true;
    }

    if(isWhite)
    {
      binaryMask[idx] = 255;
    }
    else {
      binaryMask[idx] = 0;
    }
  }
}

__device__ bool sameColor_device(unsigned short* grayMask, unsigned char* binaryMask, int rows, int cols, int idx, int movX, int movY)
{
  //Check if the adjacent location is in range
  int newIdx = idx + movX + cols * movY;
  if(newIdx >= 0 && newIdx < rows * cols)
  { 
      if(grayMask[idx] != grayMask[newIdx])
      {
        return false;
      }
      else {
        return true;
      }
  }
  else {
    return true;
  }
}


}
